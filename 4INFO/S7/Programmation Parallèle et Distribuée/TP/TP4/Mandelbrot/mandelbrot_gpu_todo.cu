
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <vector>
#include <chrono>
#include <string>
//
#include <hip/hip_complex.h>
//
#define MaxIteration 255;  //!< Le nombre max d'itération est 255, soit de base le blanc.
//
static void HandleError(	hipError_t err,
                            const char *file,
                            int line )
{
    if (err != hipSuccess)
    {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
        file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
// Cette méthode sert uniquement à sauvegarder le vecteur sous forme d'une image en niveau de gris sur 8 bits.
void save_pgm(  const char*                         filename,
                const size_t                        width,
                const size_t                        height,
                const std::vector<std::uint8_t>&   data)
{
    std::ofstream fout{ filename };
    // L'en-tête
    fout << "P2\n" << width << " " << height << " 255\n";
    for (size_t row = 0; row < height; ++row)
    {
        for (size_t col = 0; col < width; ++col)
        {
            fout << (col ? " " : "")
                 << static_cast<unsigned>(data[row * width + col]);
        }
        fout << "\n";
    }
    fout.close();
}
//
__global__ void mandel_kernel_double(std::uint8_t* img, const size_t width, const size_t height) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	const double aspect = static_cast<double>(width) / static_cast<double>(height);
	double myrow = static_cast<double>(index) / static_cast<double>(width);
	double mycol = static_cast<double>(index % width);
	double mag = 0.0;
	std::uint8_t lim = MaxIteration;
	myrow /= height;
	mycol /= width;
	hipDoubleComplex z0 = make_hipDoubleComplex(aspect * (2*mycol - 1) - 0.5, 2*myrow - 1);
	hipDoubleComplex z = make_hipDoubleComplex(0.0, 0.0);
	do {
		z = hipCadd(hipCmul(z, z), z0);
		mag = hipCabs(z);
	} while(lim-- && mag < 4.0);
	img[index] = lim;
}
//
int main(int argc, char* argv[])
{
    if (argc != 3)
    {
        std::cerr << "Usage:\n"
                  << argv[0] << " [width] [height]\n";
        return 1;
    }
    const size_t width  = std::stoul(argv[1]);
    const size_t height = std::stoul(argv[2]);
	std::uint8_t* img_GPU;
    std::vector<std::uint8_t> image(height * width, 0);
    HANDLE_ERROR(hipMalloc((void**)&img_GPU, width * height * sizeof(std::uint8_t)));
	// Note : il est possible de manipuler le pointeur de données sous-jacent au vecteur via la méthode '.data()'
    auto t0 = std::chrono::high_resolution_clock::now();
	// mandel_kernel_double<<<ceil((double)(width*height)/32.0), 32>>>(...);
    // TODO : Appeler mandel_kernel_double
    auto t1 = std::chrono::high_resolution_clock::now();
    
	std::cout << "Generation of Mandelbrot set for image size " << width << " x " << height << " took "
              << std::chrono::duration<double>(t1-t0).count() << " seconds (GPU version)\n";
    save_pgm("output_GPU.pgm", width, height, image);
}
